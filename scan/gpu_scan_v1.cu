#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include "../cuda_utility.hpp"

namespace cg = cooperative_groups;
using uint = unsigned int;
const int SECTION_LIMITATION = 1024;

/////////////////////////////////////////
// Inclusive Scan with Kogge-Stone algorithm
// Shared memory を使わずにglobal memoryですべて完結させる
/////////////////////////////////////////
__global__ void scan_gpu_kernel_v1(
  int num_elements,
  const uint* values,
  uint* prefix_sum
) {
  cg::grid_group cga = cg::this_grid();
  int global_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (global_idx < num_elements)
    prefix_sum[global_idx] = values[global_idx];
  else
    prefix_sum[global_idx] = 0;

  for (int stride = 1; stride <= global_idx; stride *= 2) {
    cg::sync(cga);
    uint temp;
    temp = prefix_sum[global_idx] + prefix_sum[global_idx - stride];
    cg::sync(cga);
    prefix_sum[global_idx] = temp;
  }
}

void launch_kernel_scan_v1(
	const int num_elements,
  uint* d_values,
  uint* d_prefix_sum,
  int num_blocks,
  int num_threads
) {
  void* args[] = {(void*)&num_elements, (void*)&d_values, (void*)&d_prefix_sum};
  checkCudaErrors(hipLaunchCooperativeKernel(
      (void*)scan_gpu_kernel_v1,
      num_blocks,
      num_threads,
      args
  ));
  hipDeviceSynchronize();
}


/////////////////////////////////////////
// Inclusive Scan with Kogge-Stone algorithm with shared memory
// block内で計算して、後で結果をまとめ上げる
// TODO:
//   * block内だけで計算する-> Sという中間出力の配列に書く
//   * まとめ上げる処理を書く（もう一度Sにscanを通す）
//   * 定数を足すカーネルを作る
/////////////////////////////////////////
__global__ void scan_gpu_kernel_v2_first_phase(
    int num_elements,
    const uint* values,
    uint* prefix_sum,
    const size_t sections_size,
    uint* sections
) {
  cg::thread_block cta = cg::this_thread_block();
  int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ uint local_prefix_sum[SECTION_LIMITATION];

  if (threadIdx.x < num_elements)
    local_prefix_sum[threadIdx.x] = values[threadIdx.x];
  else
    local_prefix_sum[threadIdx.x] = 0;

  for (uint stride = 1; stride < blockDim.x; stride *= 2) {
    cta.sync();
    uint temp;
    if (threadIdx.x >= stride)
      temp = local_prefix_sum[threadIdx.x] + local_prefix_sum[threadIdx.x - stride];
    cta.sync();
    if (threadIdx.x >= stride)
      local_prefix_sum[threadIdx.x] = temp;
  }
  if (global_idx < num_elements)
    prefix_sum[global_idx] = local_prefix_sum[threadIdx.x];

  cg::sync(cta);
  if (threadIdx.x == blockDim.x - 1) {
    sections[blockIdx.x] = local_prefix_sum[threadIdx.x];
  } 
}

__global__ void scan_gpu_kernel_v2_second_phase(
    const int num_sections,
    uint* sections
) {
  cg::thread_block cta = cg::this_thread_block();

  if (threadIdx.x >= num_sections)
    sections[threadIdx.x] = 0;
  
  for (uint stride = 1; stride <= blockDim.x; stride *= 2) {
    cta.sync();
    uint temp;
    if (threadIdx.x >= stride)
      temp = sections[threadIdx.x] + sections[threadIdx.x - stride];
    cta.sync();
    if (threadIdx.x >= stride)
      sections[threadIdx.x] = temp;
  }
}

__global__ void scan_gpu_kernel_v2_third_phase(
  const int num_elements,
  const uint* sections,
  uint* prefix_sum
) {
  int global_idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (0 < blockIdx.x && global_idx < num_elements)
    prefix_sum[global_idx] += sections[blockIdx.x - 1];
}


void launch_kernel_scan_v2(
	const int num_elements,
  uint* d_values,
  uint* d_prefix_sum,
  int num_blocks,
  int num_threads
) {
  uint* d_sections;
  size_t sections_size = num_blocks * sizeof(uint);
  checkCudaErrors(hipMalloc(&d_sections, sections_size));
  // uint* d_sections2;
  // checkCudaErrors(hipMalloc(&d_sections2, sections_size));
  // uint* d_sections_compressed;
  // uint num_sections_compressed = 
  //   (num_blocks + SECTION_LIMITATION - 1) / SECTION_LIMITATION;
  // size_t sections_compressed_size = 
  //   num_sections_compressed * sizeof(uint);
  // checkCudaErrors(hipMalloc(&d_sections_compressed, sections_compressed_size));

  // First CUDA kernel: scan on each block
  scan_gpu_kernel_v2_first_phase
    <<<num_blocks, num_threads, SECTION_LIMITATION * sizeof(uint)>>>(
      num_elements,
      d_values,
      d_prefix_sum,
      sections_size,
      d_sections
  );
  
  // Second CUDA kernel: scan on block-wise sections
  assert(num_blocks <= SECTION_LIMITATION);
  // scan_gpu_kernel_v2_first_phase
  //   <<<num_sections_compressed, num_blocks, sections_compressed_size>>>
  // (
  //     num_blocks,
  //     d_sections,
  //     d_sections_compressed,
  //     d_sections2
  // );

  scan_gpu_kernel_v2_second_phase<<<1, num_blocks>>>(
      num_blocks,
      d_sections
  );

  // Third CUDA kernel: adding a section element as a base
  scan_gpu_kernel_v2_third_phase<<<num_blocks, num_threads>>>(
      num_elements,
      d_sections,
      d_prefix_sum
  );
}
